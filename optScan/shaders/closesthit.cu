#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
/*
 * Code responsible for the closest hit of the ray shot in raygeneration.cu to the material surface. 
 * Disturbance and measurement errors apllied here.
 */


#include "app_config.h"
#include <corecrt_math.h>

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"

#define CONVERT_METER 1.e-3f
#define POLYNOMIAL_A2_CONST -5.139e-6f*1000*1000
#define POLYNOMIAL_A1_CONST 0.000992f*1000
#define POLYNOMIAL_A0_CONST 15.26f

// Context global variables provided by the renderer system.

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(MetaPerRayData, thePrd, rtPayload, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );
rtDeclareVariable(int, objectID, , );
// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute geometric_normal, );

// OptiX programming tip!
// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.
// Note that the matching between attribute outputs from the intersection program and 
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL). 
// The actual variable name doesn't need to match but it's recommended for clarity.
 

RT_PROGRAM void closest_hit_meta()
{	
  // Transform the(unnormalized) object space normals into world space.
  // Information about light and shadow in phong.cu of sutil sdk

  thePrd.worldNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal)); //Finds normal vector from the closest hit surface.

  float LIDAR_Error_Polynomial; //Equation of LIDAR sensor error model

  if (theIntersectionDistance < 60) // Maximum scanning range of chosen LiDAR sensor
  {
	  LIDAR_Error_Polynomial = POLYNOMIAL_A2_CONST *(theIntersectionDistance*theIntersectionDistance) + POLYNOMIAL_A1_CONST*(theIntersectionDistance) +POLYNOMIAL_A0_CONST; // The sensor error model in mm
	  thePrd.worldPos = theRay.origin + theRay.direction * (theIntersectionDistance + LIDAR_Error_Polynomial * CONVERT_METER + thePrd.noise*CONVERT_METER); // 3D position of the surface hit
	  thePrd.intersectDist = theIntersectionDistance + LIDAR_Error_Polynomial * CONVERT_METER + thePrd.noise*CONVERT_METER; // Total distance perceived by the virtual sensor 
																															// (including sensor error model and gaussian noise N(0,2.8)).
  }
  else //virtual sensor fails to detect outside of maximum distance range
  {
	  thePrd.worldPos = make_float3(0.0f); //sensor has no return
	  thePrd.intersectDist = 0; 

  }


  thePrd.objectID = objectID; //classification number from the 3D model hit
}
