#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>

#include "rt_function.h"
#include "per_ray_data.h"

#include <iostream>
#include <assert.h>
#include <stdio.h>


#include "random.h"

#include "rt_assert.h"
#include "helpers.h"

#include "Random123/philox.h"
#include "Random123/threefry.h"
#include "Random123/ReinterpretCtr.hpp"
#include "Random123/uniform.hpp"
#include "Random123/boxmuller.hpp"

rtBuffer<float4, 2> sysPositionsBuffer; 
rtBuffer<float4, 2> sysNormalsBuffer;	
rtBuffer<float4, 2> sysColorsBuffer;
rtBuffer<float, 2> scene_noise;


rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(uint2, theLaunchDim,   rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, meta_ray_type, , );
rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;
rtDeclareVariable(float3, angle, attribute angle, );

// Entry point for a pinhole camera.

RT_PROGRAM void raygeneration()
{

#ifdef TIME_VIEW
	clock_t t0 = clock();
#endif

	// The launch index is the pixel coordinate.
	// Note that launchIndex = (0, 0) is the bottom left corner of the image,
	// which matches the origin in the OpenGL texture used to display the result.
	const float2 pixel = make_float2(theLaunchIndex);
	// Sample the ray in the center of the pixel.
	const float2 fragment = pixel + make_float2(0.5f);
	// The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
	float2 screen = make_float2(theLaunchDim);
	//float2 screen = make_float2(256,360);
	// Normalized device coordinates in range [-1, 1].

	screen.x = fmaxf(1.0f, screen.x - 1.0f);
	screen.y = fmaxf(1.0f, screen.y - 1.0f);

	//const float2 ndc = (fragment / screen) * 2.0f - 1.0f;
	//posi��o e angula��o da vis�o esferica
	//Codigo original exemplo optix
	/*const float2 ndc = (pixel / screen) *
		make_float2(2.0f * M_PIf, M_PIf) + make_float2(M_PIf, 0);
	float3 angle = make_float3(cos(ndc.x) * sin(ndc.y),	-cos(ndc.y),sin(ndc.x) * sin(ndc.y));  */

	float totalVerticalAngle = M_PIf*((360.0f)/180.f);
	float totalHorizontalAngle = M_PIf*((300.0f)/180.f);

	/*const float2 ndc = (pixel / screen) * make_float2(totalHorizontalAngle, totalVerticalAngle) 
		- 0.5f * make_float2(totalHorizontalAngle, totalVerticalAngle);*/
	const float2 ndc = (pixel / screen) * make_float2(totalVerticalAngle, totalHorizontalAngle)
		- 0.5f * make_float2(totalVerticalAngle, totalHorizontalAngle);

	float3 angle = make_float3(
		cos(ndc.y) * sin(ndc.x), 
		sin(ndc.y),
		cos(ndc.y) * cos(ndc.x)
	);

	
	// Gerando numero aleatorio
	unsigned int seed = tea<16>(screen.x*theLaunchIndex.y + theLaunchIndex.x, ndc.x);

	typedef r123::Threefry4x64 CBRNG;

	CBRNG g;
	CBRNG::ctr_type  ctr = { { 0,6} };
	CBRNG::key_type key = { {seed} };


	ctr[0] = 4;
	CBRNG::ctr_type rand = g(ctr, key);
	
	float rand_00 = r123::u01<float>(rand[0]);
	float rand_01 = r123::u01<float>(rand[1]);
	float rand_m = (rand_00 + rand_01) * 0.5;

	float test = scene_noise[theLaunchIndex]; // testando passagem de dados do host para device

	r123::double2 ashiok = r123::boxmuller(rand[0],rand[1]);
  


	if (test > 0.5)
	{
		test = 0;
	}
	else
	{
		test = 1;
	} 

  const float3 origin   = sysCameraPosition;
//  const float3 direction = optix::normalize((ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW));
  //direction for spheric view
 const float3 direction = optix::normalize(angle.x*optix::normalize(sysCameraU) +  angle.y*optix::normalize(sysCameraV) +  angle.z*optix::normalize(sysCameraW));

  MetaPerRayData metaData;
  metaData.angle = angle;
  metaData.noise = ashiok.x*.5;
    optix::Ray ray = optix::make_Ray(origin, direction, meta_ray_type, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_object, ray, metaData);

  
  RadiancePerRayData radianceData;
  radianceData.depth = 0;
  // TODO: Assess if we can just change ray.ray_type
  //optix::Ray radianceRay = optix::make_Ray(origin, direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
  optix::Ray radianceRay = optix::make_Ray(origin, direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
  rtTrace(top_object, radianceRay, radianceData);

  //printf("P: %f, %f, %f \n U: %f, %f, %f \n V: %f, %f, %f \n W: %f, %f, %f \n I: %f\n", sysCameraPosition.x, sysCameraPosition.y, sysCameraPosition.z, sysCameraU.x, sysCameraU.y, sysCameraU.z, sysCameraV.x, sysCameraV.y, sysCameraV.z, sysCameraW.x, sysCameraW.y, sysCameraW.z, metaData.intersectDist);

#ifdef TIME_VIEW
	  clock_t t1 = clock();

  float expected_fps = 1.0f;
  float pixel_time = (t1 - t0) * time_view_scale * expected_fps;
  sysColorsBuffer[theLaunchIndex] = make_float4(make_float3(pixel_time),0);

#else
  sysPositionsBuffer[theLaunchIndex] = make_float4(metaData.worldPos, (float)metaData.objectID); //Receives 3d points and label
  //sysPositionsBuffer[theLaunchIndex] = make_float4(metaData.worldPos, (float)metaData.intersectDist);
  sysNormalsBuffer[theLaunchIndex] = make_float4(metaData.worldNormal, (float)rand_m); //Receives normal vectors
  //sysNormalsBuffer[theLaunchIndex] = make_float4(angle,(float)rand_m);
  sysColorsBuffer[theLaunchIndex] = make_float4(radianceData.result, 0); //Receives RGB color vectors
#endif
}
