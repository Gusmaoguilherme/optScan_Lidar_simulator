#include "hip/hip_runtime.h"
/*
* Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_world.h>
#include "per_ray_data.h"

using namespace optix;

rtDeclareVariable(float3, center, , );
rtDeclareVariable(float, halfHeight, , );
rtDeclareVariable(float, radius, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(RadiancePerRayData, thePrd, rtPayload, );

template<bool use_robust_method>
static __device__
void intersect_sphere(void)
{
	float3 O = ray.origin - center;
	float3 D = ray.direction;

	float b = dot(O, D);
	float c = dot(O, O) - radius * radius;
	float disc = b * b - c;
	if (disc > 0.0f)
	{
		float sdisc = sqrtf(disc);
		float root1 = (-b - sdisc);

		bool do_refine = false;

		float root11 = 0.0f;

		if (use_robust_method && fabsf(root1) > 10.f * radius)
		{
			do_refine = true;
		}

		if (do_refine)
		{
			// refine root1
			float3 O1 = O + root1 * ray.direction;
			b = dot(O1, D);
			c = dot(O1, O1) - radius * radius;
			disc = b * b - c;

			if (disc > 0.0f)
			{
				sdisc = sqrtf(disc);
				root11 = (-b - sdisc);
			}
		}

		bool check_second = true;
		if (rtPotentialIntersection(root1 + root11))
		{
			shading_normal = geometric_normal = (O + (root1 + root11)*D) / radius;
			if (rtReportIntersection(0))
				check_second = false;
		}
		if (check_second)
		{
			float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
			if (rtPotentialIntersection(root2))
			{
				shading_normal = geometric_normal = (O + root2 * D) / radius;
				rtReportIntersection(0);
			}
		}
	}
}

template<bool use_robust_method>
static __device__
void intersect_cylinder(void)
{
	float3 O = ray.origin - center;
	float3 D = ray.direction;

	float a = D.x * D.x + D.z * D.z;
	float b = 2 * (O.x * D.x + O.z * D.z);
	float c = (O.x * O.x + O.z * O.z) - radius * radius;

	float disc = b * b - 4 * a*c;

	float3 hit_p, offset;

	if (disc > 0.0f)
	{
		float root1, root2;

		float sdisc = copysign(sqrtf(disc), b);
		float q = (-b - sdisc) / 2.0;

		root1 = q / a;

		if (q != 0)
		{
			root2 = c / q;
		}
		else
		{
			root2 = root1;
		}

		if (root1 < 0) root1 = root2;
		if (root2 < 0) root2 = root1;

		float final_root = min(root1, root2);
		float3 hit_p = ray.origin + final_root * D;
		float3 dummy_normal = hit_p;

		if (abs(center.y - hit_p.y) < halfHeight)
		{
			dummy_normal.y = 0;
			dummy_normal = normalize(dummy_normal);

			if (rtPotentialIntersection(final_root))
			{
				shading_normal = geometric_normal = dummy_normal;
				rtReportIntersection(0);
			}
		}
	}
}


RT_PROGRAM void robust_intersect(int primIdx)
{
	intersect_cylinder<true>();
}

RT_PROGRAM void intersect(int primIdx)
{
	intersect_cylinder<false>();
}

RT_PROGRAM void closest_hit_radiance()
{
	thePrd.result = make_float3(1.0, 0.0, 0.0);
}			 

RT_PROGRAM void bounds(int, float result[6])
{
	const float3 rad = make_float3(radius, halfHeight, radius);

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (rad.x > 0.0f && !isinf(rad.x))
	{
		aabb->m_min = center - rad;
		aabb->m_max = center + rad;
	}
	else
	{
		aabb->invalidate();
	}
}

