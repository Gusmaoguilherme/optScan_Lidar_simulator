#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 /*
  * Code in case the ray in raygeneration.cu miss a target.
  */
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "per_ray_data.h"

rtDeclareVariable(MetaPerRayData, metaPrd, rtPayload, ); //current meta ray information

RT_PROGRAM void miss_meta()
{
  metaPrd.worldPos = make_float3(0.0f); //sensor has no return
  metaPrd.worldNormal = make_float3(0.0f);
  metaPrd.objectID = 0; // Zero value reserved to no classification
  metaPrd.intersectDist = 0;
}

rtDeclareVariable(RadiancePerRayData, radiancePrd, rtPayload, ); //current radiance ray information

RT_PROGRAM void miss_radiance()
{
	radiancePrd.result = make_float3(0.0f); // Constant black emission.
}